#include "hip/hip_runtime.h"
#include <iostream>
#include "math.h"
#include "stdlib.h"

#include "degrid_gpu.cuh"
#include "Defines.h"
#include "hip/hip_runtime.h"

//With managed memory, degrid.cpp must be compiled as CUDA
//in which case float2 and double2 are predefined.
//typedef struct {float x,y;} float2;
//typedef struct {double x,y;} double2;

#define single 77
#if PRECISION==single
#define PRECISION float
#endif

#ifndef PRECISION
#define PRECISION double
#endif
#define PASTER(x) x ## 2
#define EVALUATOR(x) PASTER(x)
#define PRECISION2 EVALUATOR(PRECISION)


void init_gcf(PRECISION2 *gcf, size_t size) {

  for (size_t sub_x=0; sub_x<8; sub_x++ )
   for (size_t sub_y=0; sub_y<8; sub_y++ )
    for(size_t x=0; x<size; x++)
     for(size_t y=0; y<size; y++) {
       //Some nonsense GCF
       PRECISION tmp = sin(6.28*x/size/8)*exp(-(1.0*x*x+1.0*y*y*sub_y)/size/size/2);
       gcf[size*size*(sub_x+sub_y*8)+x+y*size].x = tmp*sin(1.0*x*sub_x/(y+1));
       gcf[size*size*(sub_x+sub_y*8)+x+y*size].y = tmp*cos(1.0*x*sub_x/(y+1));
       //std::cout << tmp << gcf[x+y*size].x << gcf[x+y*size].y << std::endl;
     }

}

void degridCPU(PRECISION2* out, PRECISION2 *in, size_t npts, PRECISION2 *img, size_t img_dim, PRECISION2 *gcf, size_t gcf_dim) {
//degrid on the CPU
//  out (out) - the output values for each location
//  in  (in)  - the locations to be interpolated 
//  npts (in) - number of locations
//  img (in) - the image
//  img_dim (in) - dimension of the image
//  gcf (in) - the gridding convolution function
//  gcf_dim (in) - dimension of the GCF

   //offset gcf to point to the middle for cleaner code later
   gcf += GCF_DIM*(GCF_DIM+1)/2;
#pragma acc parallel loop copyout(out[0:NPOINTS]) copyin(in[0:NPOINTS],gcf[0:64*GCF_DIM*GCF_DIM],img[IMG_SIZE*IMG_SIZE]) gang
   for(size_t n=0; n<NPOINTS; n++) {
      //std::cout << "in = " << in[n].x << ", " << in[n].y << std::endl;
      int sub_x = floorf(8*(in[n].x-floorf(in[n].x)));
      int sub_y = floorf(8*(in[n].y-floorf(in[n].y)));
      //std::cout << "sub = "  << sub_x << ", " << sub_y << std::endl;
      int main_x = floor(in[n].x); 
      int main_y = floor(in[n].y); 
      //std::cout << "main = " << main_x << ", " << main_y << std::endl;
      PRECISION sum_r = 0.0;
      PRECISION sum_i = 0.0;
      #pragma acc parallel loop collapse(2) reduction(+:sum_r,sum_i) vector
      for (int a=-GCF_DIM/2; a<GCF_DIM/2 ;a++)
      for (int b=-GCF_DIM/2; b<GCF_DIM/2 ;b++) {
         PRECISION r1 = img[main_x+a+IMG_SIZE*(main_y+b)].x; 
         PRECISION i1 = img[main_x+a+IMG_SIZE*(main_y+b)].y; 
         PRECISION r2 = gcf[GCF_DIM*GCF_DIM*(8*sub_y+sub_x) + 
                        GCF_DIM*b+a].x;
         PRECISION i2 = gcf[GCF_DIM*GCF_DIM*(8*sub_y+sub_x) + 
                        GCF_DIM*b+a].y;
         //std::cout << r1 << std::endl;
         //std::cout << i1 << std::endl;
         //std::cout << r2 << std::endl;
         //std::cout << i2 << std::endl;
         sum_r += r1*r2 - i1*i2; 
         sum_i += r1*i2 + r2*i1;
      }
      out[n].x = sum_r;
      out[n].y = sum_i;
      //std::cout << "val = " << out[n].r << "+ i" << out[n].i << std::endl;
   } 
   gcf -= GCF_DIM*(GCF_DIM+1)/2;
}
int main(void) {

#ifdef __MANAGED
   PRECISION2* out, *in, *img, *gcf;
   hipMallocManaged(&out, sizeof(PRECISION2)*NPOINTS);
   hipMallocManaged(&in, sizeof(PRECISION2)*NPOINTS);
   hipMallocManaged(&img, sizeof(PRECISION2)*(IMG_SIZE*IMG_SIZE+2*IMG_SIZE*GCF_DIM+2*GCF_DIM));
   hipMallocManaged(&gcf, sizeof(PRECISION2)*64*GCF_DIM*GCF_DIM);
#else
   PRECISION2* out = (PRECISION2*) malloc(sizeof(PRECISION2)*NPOINTS);
   PRECISION2* in = (PRECISION2*) malloc(sizeof(PRECISION2)*NPOINTS);
   PRECISION2 *img = (PRECISION2*) malloc((IMG_SIZE*IMG_SIZE+2*IMG_SIZE*GCF_DIM+2*GCF_DIM)*sizeof(PRECISION2));

   PRECISION2 *gcf = (PRECISION2*) malloc(64*GCF_DIM*GCF_DIM*sizeof(PRECISION2));
#endif

   //img is padded (above and below) to avoid overruns
   img += IMG_SIZE*GCF_DIM+GCF_DIM;
    
   init_gcf(gcf, GCF_DIM);
   srand(2541617);
   for(size_t n=0; n<NPOINTS; n++) {
      in[n].x = ((float)rand())/RAND_MAX*1000;
      in[n].y = ((float)rand())/RAND_MAX*1000;
   }
   for(size_t x=0; x<IMG_SIZE;x++)
   for(size_t y=0; y<IMG_SIZE;y++) {
      img[x+IMG_SIZE*y].x = exp(-((x-1400.0)*(x-1400.0)+(y-3800.0)*(y-3800.0))/8000000.0)+1.0;
      img[x+IMG_SIZE*y].y = 0.4;
   }
   //Zero the data in the offset areas
   for (int x=-IMG_SIZE*GCF_DIM-GCF_DIM;x<0;x++) {
      img[x].x = 0.0; img[x].y = 0.0;
   }
   for (int x=0;x<IMG_SIZE*GCF_DIM+GCF_DIM;x++) {
      img[x+IMG_SIZE*IMG_SIZE].x = 0.0; img[x+IMG_SIZE*IMG_SIZE].y = 0.0;
   }

   degridGPU(out,in,NPOINTS,img,IMG_SIZE,gcf,GCF_DIM);
#ifdef __CPU_CHECK
   PRECISION2 *out_cpu=(PRECISION2*)malloc(sizeof(PRECISION2)*NPOINTS);
   degridCPU(out_cpu,in,NPOINTS,img,IMG_SIZE,gcf,GCF_DIM);
#endif


#ifdef __CPU_CHECK
   for (size_t n = 0; n < NPOINTS; n++) {
     if (fabs(out[n].x-out_cpu[n].x) > 0.0000001 ||
         fabs(out[n].y-out_cpu[n].y) > 0.0000001 )
        std::cout << "F(" << in[n].x << ", " << in[n].y << ") = " 
                  << out[n].x << ", " << out[n].y 
                  << " vs. " << out_cpu[n].x << ", " << out_cpu[n].y 
                  << std::endl;
   }
#endif
   img -= GCF_DIM + IMG_SIZE*GCF_DIM;
#ifdef __MANAGED
   hipFree(out);
   hipFree(in);
   hipFree(img);
   hipFree(gcf);
#else
   free(out);
   free(in);
   free(img);
   free(gcf);
#endif
}
