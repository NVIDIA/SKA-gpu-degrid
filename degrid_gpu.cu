#include "hip/hip_runtime.h"
#include "Defines.h"
#include "cucommon.cuh"
#include <iostream>

void CUDA_CHECK_ERR(unsigned lineNumber, const char* fileName) {

   hipError_t err = hipGetLastError();
   if (err) std::cout << "Error " << err << " on line " << lineNumber << " of " << fileName << ": " << hipGetErrorString(err) << std::endl;
}

float getElapsed(hipEvent_t start, hipEvent_t stop) {
   float elapsed;
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed, start, stop);
   return elapsed;
}
__device__ int2 convert(int asize, int Qpx, float pin) {

   float frac; float round;
   //TODO add the 1 afterward?
   frac = modf((pin+1)*asize, &round);
   return make_int2(int(round), int(frac*Qpx));
}

__device__ double make_zero(double2* in) { return (double)0.0;}
__device__ float make_zero(float2* in) { return (float)0.0;}

template <int gcf_dim, class CmplxType>
__global__ void degrid_kernel(CmplxType* out, CmplxType* in, size_t npts, CmplxType* img, 
                              size_t img_dim, CmplxType* gcf) {
   
   __shared__ CmplxType shm[1024/gcf_dim][gcf_dim+1];
   __shared__ CmplxType inbuff[32];
   for (int n = 32*blockIdx.x; n<npts; n+= 32*gridDim.x) {
   if (threadIdx.y == 0 && threadIdx.x<32) inbuff[threadIdx.x] = in[n+threadIdx.x];
   __syncthreads();
   for (int q=0;q<32;q++) {
      CmplxType inn = inbuff[q];
      int sub_x = floorf(GCF_GRID*(inn.x-floorf(inn.x)));
      int sub_y = floorf(GCF_GRID*(inn.y-floorf(inn.y)));
      int main_x = floorf(inn.x); 
      int main_y = floorf(inn.y); 
      auto sum_r = make_zero(img);
      auto sum_i = make_zero(img);
      int a = threadIdx.x-gcf_dim/2;
      for(int b = threadIdx.y-gcf_dim/2;b<gcf_dim/2;b+=blockDim.y)
      {
         auto r1 = img[main_x+a+img_dim*(main_y+b)].x; 
         auto i1 = img[main_x+a+img_dim*(main_y+b)].y; 
         auto r2 = __ldg(&gcf[gcf_dim*gcf_dim*(GCF_GRID*sub_y+sub_x) + 
                        gcf_dim*b+a].x);
         auto i2 = __ldg(&gcf[gcf_dim*gcf_dim*(GCF_GRID*sub_y+sub_x) + 
                        gcf_dim*b+a].y);
         sum_r += r1*r2 - i1*i2; 
         sum_i += r1*i2 + r2*i1;
      }

      //reduce in two directions
      //WARNING: Adjustments must be made if blockDim.y and blockDim.x are no
      //         powers of 2 
      shm[threadIdx.y][threadIdx.x].x = sum_r;
      shm[threadIdx.y][threadIdx.x].y = sum_i;
      __syncthreads();
      //Reduce in y
      for(int s = blockDim.y/2;s>0;s/=2) {
         if (threadIdx.y < s) {
           shm[threadIdx.y][threadIdx.x].x += shm[threadIdx.y+s][threadIdx.x].x;
           shm[threadIdx.y][threadIdx.x].y += shm[threadIdx.y+s][threadIdx.x].y;
         }
         __syncthreads();
      }

      //Reduce the top row
      for(int s = blockDim.x/2;s>16;s/=2) {
         if (0 == threadIdx.y && threadIdx.x < s) 
                    shm[0][threadIdx.x].x += shm[0][threadIdx.x+s].x;
         if (0 == threadIdx.y && threadIdx.x < s) 
                    shm[0][threadIdx.x].y += shm[0][threadIdx.x+s].y;
         __syncthreads();
      }
      if (threadIdx.y == 0) {
         //Reduce the final warp using shuffle
         CmplxType tmp = shm[0][threadIdx.x];
         for(int s = blockDim.x < 16 ? blockDim.x : 16; s>0;s/=2) {
            tmp.x += __shfl_down(tmp.x,s);
            tmp.y += __shfl_down(tmp.y,s);
         }
         
         if (threadIdx.x == 0) {
            out[n+q] = tmp;
         }
      }
   }
   }
}

template <class CmplxType>
void degridGPU(CmplxType* out, CmplxType* in, size_t npts, CmplxType *img, size_t img_dim, 
               CmplxType *gcf, size_t gcf_dim) {
//degrid on the GPU
//  out (out) - the output values for each location
//  in  (in)  - the locations to be interpolated 
//  npts (in) - number of locations
//  img (in) - the image
//  img_dim (in) - dimension of the image
//  gcf (in) - the gridding convolution function
//  gcf_dim (in) - dimension of the GCF

   CmplxType *d_out, *d_in, *d_img, *d_gcf;

   hipEvent_t start, stop;
   hipEventCreate(&start); hipEventCreate(&stop);

   CUDA_CHECK_ERR(__LINE__,__FILE__);
#ifdef __MANAGED
   d_img = img;
   d_gcf = gcf;
   d_out = out;
   d_in = in;
   std::cout << "img size = " << (img_dim*img_dim+2*img_dim*gcf_dim+2*gcf_dim)*
                                                                 sizeof(CmplxType) << std::endl;
   std::cout << "out size = " << sizeof(CmplxType)*npts << std::endl;
#else
   //img is padded to avoid overruns. Subtract to find the real head
   img -= img_dim*gcf_dim+gcf_dim;

   //Allocate GPU memory
   std::cout << "img size = " << (img_dim*img_dim+2*img_dim*gcf_dim+2*gcf_dim)*
                                                                 sizeof(CmplxType) << std::endl;
   hipMalloc(&d_img, sizeof(CmplxType)*(img_dim*img_dim+2*img_dim*gcf_dim+2*gcf_dim));
   hipMalloc(&d_gcf, sizeof(CmplxType)*64*gcf_dim*gcf_dim);
   hipMalloc(&d_out, sizeof(CmplxType)*npts);
   hipMalloc(&d_in, sizeof(CmplxType)*npts);
   std::cout << "out size = " << sizeof(CmplxType)*npts << std::endl;
   CUDA_CHECK_ERR(__LINE__,__FILE__);

   //Copy in img, gcf and out
   hipEventRecord(start);
   hipMemcpy(d_img, img, 
              sizeof(CmplxType)*(img_dim*img_dim+2*img_dim*gcf_dim+2*gcf_dim), 
              hipMemcpyHostToDevice);
   hipMemcpy(d_gcf, gcf, sizeof(CmplxType)*64*gcf_dim*gcf_dim, 
              hipMemcpyHostToDevice);
   hipMemcpy(d_in, in, sizeof(CmplxType)*npts,
              hipMemcpyHostToDevice);
   CUDA_CHECK_ERR(__LINE__,__FILE__);
   std::cout << "memcpy time: " << getElapsed(start, stop) << std::endl;

   //move d_img and d_gcf to remove padding
   d_img += img_dim*gcf_dim+gcf_dim;
#endif
   //offset gcf to point to the middle of the first GCF for cleaner code later
   d_gcf += gcf_dim*(gcf_dim+1)/2;

   hipEventRecord(start);
   degrid_kernel<128>
            <<<npts/32,dim3(gcf_dim,512/gcf_dim)>>>(d_out,d_in,npts,d_img,img_dim,d_gcf); 
   float kernel_time = getElapsed(start,stop);
   std::cout << "Processed " << npts << " complex points in " << kernel_time << " ms." << std::endl;
   std::cout << npts / 1000000.0 / kernel_time * gcf_dim * gcf_dim * 8 << "Gflops" << std::endl;
   CUDA_CHECK_ERR(__LINE__,__FILE__);

#ifdef __MANAGED
   hipDeviceSynchronize();
#else
   hipMemcpy(out, d_out, sizeof(CmplxType)*npts, hipMemcpyDeviceToHost);
   CUDA_CHECK_ERR(__LINE__,__FILE__);

   //Restore d_img and d_gcf for deallocation
   d_img -= img_dim*gcf_dim+gcf_dim;
   d_gcf -= gcf_dim*(gcf_dim+1)/2;
   hipFree(d_out);
   hipFree(d_img);
#endif
   hipEventDestroy(start); hipEventDestroy(stop);
   CUDA_CHECK_ERR(__LINE__,__FILE__);
}
template void degridGPU<double2>(double2* out, double2* in, size_t npts, double2 *img, 
                                 size_t img_dim, double2 *gcf, size_t gcf_dim); 
template void degridGPU<float2>(float2* out, float2* in, size_t npts, float2 *img, 
                                size_t img_dim, float2 *gcf, size_t gcf_dim); 
