#include "hip/hip_runtime.h"
#include "Defines.h"
#include "cucommon.cuh"
#include <iostream>

void CUDA_CHECK_ERR(unsigned lineNumber, const char* fileName) {

   hipError_t err = hipGetLastError();
   if (err) std::cout << "Error " << err << " on line " << lineNumber << " of " << fileName << ": " << hipGetErrorString(err) << std::endl;
}

float getElapsed(hipEvent_t start, hipEvent_t stop) {
   float elapsed;
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed, start, stop);
   return elapsed;
}
template <class CmplxType>
__global__ void degrid_kernel(CmplxType* out, CmplxType* in, CmplxType* img, CmplxType* gcf, size_t npts) {
   
   __shared__ CmplxType shm[1024/GCF_DIM][GCF_DIM+1];
   for (int n = blockIdx.x; n<npts; n+= gridDim.x) {
      int sub_x = floorf(8*(in[n].x-floorf(in[n].x)));
      int sub_y = floorf(8*(in[n].y-floorf(in[n].y)));
      int main_x = floorf(in[n].x); 
      int main_y = floorf(in[n].y); 
      auto sum_r = img[0].x * 0.0;
      auto sum_i = sum_r;
      int a = threadIdx.x-GCF_DIM/2;
      for(int b = threadIdx.y-GCF_DIM/2;b<GCF_DIM/2;b+=blockDim.y)
      {
         auto r1 = img[main_x+a+IMG_SIZE*(main_y+b)].x; 
         auto i1 = img[main_x+a+IMG_SIZE*(main_y+b)].y; 
         auto r2 = gcf[GCF_DIM*GCF_DIM*(8*sub_y+sub_x) + 
                        GCF_DIM*b+a].x;
         auto i2 = gcf[GCF_DIM*GCF_DIM*(8*sub_y+sub_x) + 
                        GCF_DIM*b+a].y;
         sum_r += r1*r2 - i1*i2; 
         sum_i += r1*i2 + r2*i1;
      }

      //reduce in two directions
      //WARNING: Adjustments must be made if blockDim.y and blockDim.x are no
      //         powers of 2 
      shm[threadIdx.y][threadIdx.x].x = sum_r;
      shm[threadIdx.y][threadIdx.x].y = sum_i;
      __syncthreads();
      //Reduce in y
      for(int s = blockDim.y/2;s>0;s/=2) {
         if (threadIdx.y < s) {
           shm[threadIdx.y][threadIdx.x].x += shm[threadIdx.y+s][threadIdx.x].x;
           shm[threadIdx.y][threadIdx.x].y += shm[threadIdx.y+s][threadIdx.x].y;
         }
         __syncthreads();
         if (s==1) break;
      }

      //Reduce the top row
      if (threadIdx.y > 0) continue;
      for(int s = blockDim.x/2;s>16;s/=2) {
         if (threadIdx.x < s) shm[0][threadIdx.x].x += shm[0][threadIdx.x+s].x;
         if (threadIdx.x < s) shm[0][threadIdx.x].y += shm[0][threadIdx.x+s].y;
         __syncthreads();
      }
      //Reduce the final warp using shuffle
      CmplxType tmp = shm[0][threadIdx.x];
      for(int s = blockDim.x < 16 ? blockDim.x : 16; s>0;s/=2) {
         tmp.x += __shfl_down(tmp.x,s);
         tmp.y += __shfl_down(tmp.y,s);
      }
         
      if (threadIdx.x == 0) {
         out[n] = tmp;
      }
   }
}

template <class CmplxType>
void degridGPU(CmplxType* out, CmplxType* in, CmplxType *img, CmplxType *gcf) {
//degrid on the CPU
//  out (inout) - the locations to be interpolated
//  img (in) - the image
//  gcf (in) - the gridding convolution function
   CmplxType *d_out, *d_in, *d_img, *d_gcf;

   hipEvent_t start, stop;
   hipEventCreate(&start); hipEventCreate(&stop);

   CUDA_CHECK_ERR(__LINE__,__FILE__);
   //img is padded to avoid overruns. Subtract to find the real head
   img -= IMG_SIZE*GCF_DIM+GCF_DIM;

   //Allocate GPU memory
   std::cout << "img size = " << (IMG_SIZE*IMG_SIZE+2*IMG_SIZE*GCF_DIM+2*GCF_DIM)*sizeof(CmplxType) << std::endl;
   hipMalloc(&d_img, sizeof(CmplxType)*(IMG_SIZE*IMG_SIZE+2*IMG_SIZE*GCF_DIM+2*GCF_DIM));
   hipMalloc(&d_gcf, sizeof(CmplxType)*64*GCF_DIM*GCF_DIM);
   hipMalloc(&d_out, sizeof(CmplxType)*NPOINTS);
   hipMalloc(&d_in, sizeof(CmplxType)*NPOINTS);
   std::cout << "out size = " << sizeof(CmplxType)*NPOINTS << std::endl;
   CUDA_CHECK_ERR(__LINE__,__FILE__);

   //Copy in img, gcf and out
   hipEventRecord(start);
   hipMemcpy(d_img, img, 
              sizeof(CmplxType)*(IMG_SIZE*IMG_SIZE+2*IMG_SIZE*GCF_DIM+2*GCF_DIM), 
              hipMemcpyHostToDevice);
   CUDA_CHECK_ERR(__LINE__,__FILE__);
   hipMemcpy(d_gcf, gcf, sizeof(CmplxType)*64*GCF_DIM*GCF_DIM, 
              hipMemcpyHostToDevice);
   CUDA_CHECK_ERR(__LINE__,__FILE__);
   hipMemcpy(d_in, in, sizeof(CmplxType)*NPOINTS,
              hipMemcpyHostToDevice);
   CUDA_CHECK_ERR(__LINE__,__FILE__);
   std::cout << "memcpy time: " << getElapsed(start, stop) << std::endl;

   //move d_img and d_gcf to remove padding
   d_img += IMG_SIZE*GCF_DIM+GCF_DIM;
   //offset gcf to point to the middle of the first GCF for cleaner code later
   d_gcf += GCF_DIM*(GCF_DIM+1)/2;

   hipEventRecord(start);
   degrid_kernel<<<NPOINTS,dim3(GCF_DIM,1024/GCF_DIM)>>>(d_out,d_in,d_img,d_gcf,NPOINTS); 
   float kernel_time = getElapsed(start,stop);
   std::cout << "kernel time: " << kernel_time << std::endl;
   std::cout << NPOINTS / 1000000.0 / kernel_time * GCF_DIM * GCF_DIM * 8 << "Gflops" << std::endl;
   CUDA_CHECK_ERR(__LINE__,__FILE__);

   hipMemcpy(out, d_out, sizeof(CmplxType)*NPOINTS, hipMemcpyDeviceToHost);
   CUDA_CHECK_ERR(__LINE__,__FILE__);

   //Restore d_img and d_gcf for deallocation
   d_img -= IMG_SIZE*GCF_DIM+GCF_DIM;
   d_gcf -= GCF_DIM*(GCF_DIM+1)/2;
   hipFree(d_out);
   hipFree(d_img);
   hipEventDestroy(start); hipEventDestroy(stop);
   CUDA_CHECK_ERR(__LINE__,__FILE__);
}
template void degridGPU<double2>(double2* out, double2* in, double2 *img, double2 *gcf); 
template void degridGPU<float2>(float2* out, float2* in, float2 *img, float2 *gcf); 
